#include "hip/hip_runtime.h"
// HEADER FILES
#include <hip/hip_runtime.h>
#include <stdio.h>

// ERROR HANDING MACRO
#define CHECK(call)                                                                     \
{											\
	const cudaError_t_error = call;							\
	if ( error != hipSuccess )							\
	{										\
		printf("Error: %s:%d, ", __FILE__, __LINE__ );				\
		printf("Code: %d,reason: %s \n ",error, hipGetErrorString( error ));	\
		exit(1);								\
	}										\
}

// VERIFICATION OF KERNEL 
void checkResult( float *hostRef, float *gpuRef, const int N ) 
{
	double epsilon		=	1.0E-8;
	bool match		=	1;
	for( int i = 0; i < N; i++ )
	{
		if ( abs( hostRef[i] - gpuRef[i] > epsilon ) )
		{
			match	=	0;
			printf(" Arrays do not match ! \n ");
			printf(" host %5.2f gpu %5.2f at current %d \n ", hostRef[i],gpuRef[i],i);	
			break;
		}
	} 
	if( match ) printf("Arrays do match ! \n \n ");
}

// INITIALISING DATA 
void initialData( float *ip, int size ) 
{
	time_t t;
	srand( (unsigned) time(&t) );
	// GENERATE DIFFERENT SEED FOR RANDOM NUMBER
	
	for( int i = 0; i < size; i++ )
	{
		ip[i]	= 	(float) ( rand() & 0XFF ) / 10.0f;
	}
}

void sumArraysOnHost( float *A, float *B, float *C, const int N )
{
	for( int idx = 0; idx < N; idx++ )
	C[idx]	=	A[idx] + B[idx];
}

__global__ void sumArraysOnGPU( float *A, float *B, float *C) 
{
	int i	=	blockIdx.x * blockDim.x + threadIdx.x;
	C[i]	=	A[i] + B[i];
	printf("threadIdx: (%d,%d,%d) | blockIdx:(%d,%d,%d)| blockDim: (%d,%d,%d)| gridDim:(%d,%d,%d)|    \n Array location : %d \n",threadIdx.x,threadIdx.y,threadIdx.z, blockIdx.x,blockIdx.y,blockIdx.z, blockDim.x,blockDim.y,blockDim.z,gridDim.x,gridDim.y,gridDim.z,i);
}

int main( int argc, char **argv )
{
	printf(" %s Starting... \n", argv[0]);

	// SET UP DEVICE
	int dev		= 0;
	hipSetDevice(dev);

	// SET UP DATA SIZE OF VECTORS
	int nElem	= 64;
	printf("Vector size %d \n ", nElem);

	// MALLOC HOST MEMORY
	size_t nBytes	= nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A		= (float *) malloc(nBytes);
	h_B		= (float *) malloc(nBytes);
	hostRef		= (float *) malloc(nBytes);
	gpuRef		= (float *) malloc(nBytes);

	// INITIALIZE DATA FROM HOST SIDE
	initialData( h_A, nElem );
	initialData( h_B, nElem );

	memset( hostRef, 0, nBytes);
	memset(  gpuRef, 0, nBytes);

	// MALLOC DEVICE GLOBAL MEMORY
	float *d_A, *d_B, *d_C;
	hipMalloc( (float**)&d_A, nBytes);
	hipMalloc( (float**)&d_B, nBytes);
	hipMalloc( (float**)&d_C, nBytes);

	// TRANSFER DATA FROM HOST TO DEVICE
	hipMemcpy( d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy( d_B, h_B, nBytes, hipMemcpyHostToDevice);

	// INVOKE KERNEL AT HOST SIDE
	// dim3 block ( nElem );
	dim3 block ( 16 );
	dim3 grid  ( nElem / block.x );

	sumArraysOnGPU<<< grid, block >>>( d_A, d_B, d_C );
	printf("Execution configuration <<< %d, %d >>> \n ",grid.x,block.x);

	// COPY KERNEL RESULT BACK TO HOST SIDE
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// ADD VECTOR FROM HOST SIDE FOR CHECKING 
	sumArraysOnHost( h_A, h_B, hostRef, nElem);

	// CHECK DEVICE RESULTS
	checkResult( hostRef, gpuRef, nElem);

	// FREE DEVICE GLOBAL MEMORY
	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

	// FREE HOST MEMORY
	free( h_A );
	free( h_B );
	free( hostRef );
	free( gpuRef );
	
	return(0);
}

